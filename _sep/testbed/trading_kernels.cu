#include <hip/hip_runtime.h>

namespace sep { namespace testbed {

namespace {
__global__ void scaleBiasKernel(const float* input, float* output, int n, float scale, float bias) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx] = input[idx] * scale + bias;
    }
}

inline hipError_t launchScaleBias(const float* input, float* output, int n, float scale, float bias) {
    if (!input || !output || n <= 0) return hipErrorInvalidValue;
    dim3 blockSize(256);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x);
    scaleBiasKernel<<<gridSize, blockSize>>>(input, output, n, scale, bias);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) return err;
    return hipDeviceSynchronize();
}
} // anonymous namespace

hipError_t analyzePatterns(const float* market_data, float* analysis_results, int data_points) {
    return launchScaleBias(market_data, analysis_results, data_points, 0.8f, 0.0f);
}

hipError_t trainQuantumPatterns(const float* input_data, float* output_patterns, int data_size, int /*pattern_count*/) {
    return launchScaleBias(input_data, output_patterns, data_size, 0.5f, 0.5f);
}

}} // namespace sep::testbed

