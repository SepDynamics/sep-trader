#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void oldPatternKernel(const float* input, float* output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float val = input[idx];
        #pragma unroll
        for (int i = 0; i < 3; ++i) {
            val = val * 0.8f + 0.1f; // extra work
        }
        output[idx] = val;
    }
}

__global__ void oldQuantumKernel(const float* input, float* output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float val = input[idx];
        #pragma unroll
        for (int i = 0; i < 3; ++i) {
            val = val * 0.5f + 0.5f; // extra work
        }
        output[idx] = val;
    }
}

__global__ void scaleBiasKernel(const float* input, float* output, int n, float scale, float bias) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx] = input[idx] * scale + bias;
    }
}

float benchmarkOld(const float* d_input, float* d_output, int n) {
    dim3 blockSize(256);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    oldPatternKernel<<<gridSize, blockSize>>>(d_input, d_output, n);
    oldQuantumKernel<<<gridSize, blockSize>>>(d_input, d_output, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return ms;
}

float benchmarkNew(const float* d_input, float* d_output, int n) {
    dim3 blockSize(256);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    scaleBiasKernel<<<gridSize, blockSize>>>(d_input, d_output, n, 0.8f, 0.0f);
    scaleBiasKernel<<<gridSize, blockSize>>>(d_input, d_output, n, 0.5f, 0.5f);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return ms;
}

int main() {
    const int N = 1 << 20; // ~1M elements
    std::vector<float> h_input(N, 1.0f);
    float *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));
    hipMemcpy(d_input, h_input.data(), N * sizeof(float), hipMemcpyHostToDevice);

    float oldTime = benchmarkOld(d_input, d_output, N);
    float newTime = benchmarkNew(d_input, d_output, N);

    std::cout << "old(ms):" << oldTime << " new(ms):" << newTime
              << " speedup:" << oldTime / newTime << "x" << std::endl;

    hipFree(d_input);
    hipFree(d_output);
    return 0;
}

