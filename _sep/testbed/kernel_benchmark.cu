#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

namespace sep { namespace testbed {
hipError_t processMultiPair(
    const float* pair_data,
    float* processed_signals,
    int pair_count,
    int data_per_pair);
}} // namespace sep::testbed

__global__ void oldPatternKernel(const float* input, float* output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float val = input[idx];
        #pragma unroll
        for (int i = 0; i < 3; ++i) {
            val = val * 0.8f + 0.1f; // extra work
        }
        output[idx] = val;
    }
}

__global__ void oldQuantumKernel(const float* input, float* output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float val = input[idx];
        #pragma unroll
        for (int i = 0; i < 3; ++i) {
            val = val * 0.5f + 0.5f; // extra work
        }
        output[idx] = val;
    }
}

__global__ void scaleBiasKernel(const float* input, float* output, int n, float scale, float bias) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx] = input[idx] * scale + bias;
    }
}

float benchmarkOld(const float* d_input, float* d_output, int n) {
    dim3 blockSize(256);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    oldPatternKernel<<<gridSize, blockSize>>>(d_input, d_output, n);
    oldQuantumKernel<<<gridSize, blockSize>>>(d_input, d_output, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return ms;
}

__global__ void oldMultiPairKernel(const float* pair_data, float* processed, int pair_count, int data_per_pair) {
    int pair_idx = blockIdx.x;
    int data_idx = threadIdx.x;
    if (pair_idx < pair_count && data_idx < data_per_pair) {
        int global_idx = pair_idx * data_per_pair + data_idx;
        processed[global_idx] = pair_data[global_idx] * 0.9f + 0.1f;
    }
}

float benchmarkOldMultiPair(const float* pair_data, float* processed, int pair_count, int data_per_pair) {
    dim3 blockSize(data_per_pair);
    dim3 gridSize(pair_count);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    oldMultiPairKernel<<<gridSize, blockSize>>>(pair_data, processed, pair_count, data_per_pair);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return ms;
}

float benchmarkNewMultiPair(const float* pair_data, float* processed, int pair_count, int data_per_pair) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    sep::testbed::processMultiPair(pair_data, processed, pair_count, data_per_pair);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return ms;
}

float benchmarkNew(const float* d_input, float* d_output, int n) {
    dim3 blockSize(256);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    scaleBiasKernel<<<gridSize, blockSize>>>(d_input, d_output, n, 0.8f, 0.0f);
    scaleBiasKernel<<<gridSize, blockSize>>>(d_input, d_output, n, 0.5f, 0.5f);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return ms;
}

int main() {
    const int N = 1 << 20; // ~1M elements
    std::vector<float> h_input(N, 1.0f);
    float *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));
    hipMemcpy(d_input, h_input.data(), N * sizeof(float), hipMemcpyHostToDevice);

    float oldTime = benchmarkOld(d_input, d_output, N);
    float newTime = benchmarkNew(d_input, d_output, N);

    std::cout << "old(ms):" << oldTime << " new(ms):" << newTime
              << " speedup:" << oldTime / newTime << "x" << std::endl;

    const int pairs = 32;
    const int per_pair = 256;
    size_t bytes = pairs * per_pair * sizeof(float);
    float *d_pair, *d_proc;
    hipMalloc(&d_pair, bytes);
    hipMalloc(&d_proc, bytes);
    hipMemcpy(d_pair, h_input.data(), bytes, hipMemcpyHostToDevice);

    float oldMulti = benchmarkOldMultiPair(d_pair, d_proc, pairs, per_pair);
    float newMulti = benchmarkNewMultiPair(d_pair, d_proc, pairs, per_pair);
    std::cout << "old_multi(ms):" << oldMulti << " new_multi(ms):" << newMulti
              << " speedup:" << oldMulti / newMulti << "x" << std::endl;

    hipFree(d_pair);
    hipFree(d_proc);

    hipFree(d_input);
    hipFree(d_output);
    return 0;
}

