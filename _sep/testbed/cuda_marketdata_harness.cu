#include "hip/hip_runtime.h"
#include "cuda_marketdata_harness.hpp"
#include <hip/hip_runtime.h>
#include <stdexcept>

__global__ void doubleKernel(const double* in, double* out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = in[idx] * 2.0;
    }
}

std::vector<double> cpuDoubleMid(const std::vector<sep::connectors::MarketData>& data) {
    std::vector<double> out;
    out.reserve(data.size());
    for (const auto& md : data) {
        out.push_back(md.mid * 2.0);
    }
    return out;
}

std::vector<double> gpuDoubleMid(const std::vector<sep::connectors::MarketData>& data) {
    int n = static_cast<int>(data.size());
    std::vector<double> in(n);
    for (int i = 0; i < n; ++i) {
        in[i] = data[i].mid;
    }

    double *d_in = nullptr, *d_out = nullptr;
    size_t bytes = n * sizeof(double);
    hipError_t err;
    err = hipMalloc(&d_in, bytes);
    if (err != hipSuccess) {
        throw std::runtime_error("hipMalloc d_in failed");
    }
    err = hipMalloc(&d_out, bytes);
    if (err != hipSuccess) {
        hipFree(d_in);
        throw std::runtime_error("hipMalloc d_out failed");
    }

    err = hipMemcpy(d_in, in.data(), bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(d_in);
        hipFree(d_out);
        throw std::runtime_error("hipMemcpy to device failed");
    }

    int block = 256;
    int grid = (n + block - 1) / block;
    doubleKernel<<<grid, block>>>(d_in, d_out, n);
    err = hipGetLastError();
    if (err != hipSuccess) {
        hipFree(d_in);
        hipFree(d_out);
        throw std::runtime_error("Kernel launch failed");
    }

    std::vector<double> out(n);
    err = hipMemcpy(out.data(), d_out, bytes, hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
    if (err != hipSuccess) {
        throw std::runtime_error("hipMemcpy to host failed");
    }

    return out;
}

