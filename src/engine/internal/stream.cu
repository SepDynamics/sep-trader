#include <hip/hip_runtime.h>

#include <stdexcept>

#include "stream.h"

namespace sep {
namespace cuda {

Stream::Stream() {
    if (hipStreamCreate(&stream_) != hipSuccess) {
        throw std::runtime_error("Failed to create CUDA stream");
    }
}

Stream::~Stream() {
    if (stream_) {
        hipStreamDestroy(stream_);
    }
}

Stream::Stream(Stream&& other) noexcept : stream_(other.stream_) {
    other.stream_ = nullptr;
}

Stream& Stream::operator=(Stream&& other) noexcept {
    if (this != &other) {
        if (stream_) {
            hipStreamDestroy(stream_);
        }
        stream_ = other.stream_;
        other.stream_ = nullptr;
    }
    return *this;
}

void Stream::synchronize() {
    if (stream_) {
        hipStreamSynchronize(stream_);
    }
}

void Stream::wait(hipEvent_t event) {
    if (stream_) {
        hipStreamWaitEvent(stream_, event, 0);
    }
}

void Stream::record(hipEvent_t event) {
    if (stream_) {
        hipEventRecord(event, stream_);
    }
}

hipStream_t Stream::handle() const {
    return stream_;
}

bool Stream::isValid() const {
    return stream_ != nullptr;
}

std::shared_ptr<Stream> Stream::create(unsigned int flags) {
    auto stream = std::make_shared<Stream>();
    if (hipStreamCreateWithFlags(&stream->stream_, flags) != hipSuccess) {
        throw std::runtime_error("Failed to create CUDA stream");
    }
    return stream;
}

}  // namespace cuda
}  // namespace sep
