#include "hip/hip_runtime.h"
// CRITICAL: For CUDA compilation, include our special CUDA header fix first
#include "util/global_includes.h"
#include "cuda_prerequisites.h"

// This file now includes all necessary headers, including <hip/hip_runtime.h>

// GLM CUDA compatibility
#include <glm/glm.hpp>

#include "cuda_math_compat.h"
#include "quantum_processor_cuda.h"

namespace sep::quantum {

// A simple CUDA kernel to calculate coherence (dot product)
__global__ void coherenceKernel(const float* a, const float* b, float* result) {
    int i = threadIdx.x;
    result[i] = a[i] * b[i];
}

QuantumProcessorCUDA::QuantumProcessorCUDA(const Config& config) : QuantumProcessor(config) {
    // Constructor implementation
}

QuantumProcessorCUDA::~QuantumProcessorCUDA() {
    // Destructor implementation
}

float QuantumProcessorCUDA::calculateCoherence(const glm::vec3& a, const glm::vec3& b) {
    // This is a simplified example. A real implementation would handle batches of vectors.
    float result = 0.0f;
    
    // Convert glm::vec3 to float arrays
    float h_a[3] = {a.x, a.y, a.z};
    float h_b[3] = {b.x, b.y, b.z};
    
    // Allocate memory on the device
    float* d_a;
    float* d_b;
    float* d_result;
    hipMalloc((void**)&d_a, sizeof(float) * 3);
    hipMalloc((void**)&d_b, sizeof(float) * 3);
    hipMalloc((void**)&d_result, sizeof(float) * 3);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, sizeof(float) * 3, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float) * 3, hipMemcpyHostToDevice);

    // Launch the kernel
    coherenceKernel<<<1, 3>>>(d_a, d_b, d_result);

    // Copy result back from device to host
    float h_result[3];
    hipMemcpy(h_result, d_result, sizeof(float) * 3, hipMemcpyDeviceToHost);

    // Sum the partial results and normalize
    float dot_product = 0.0f;
    for (int i = 0; i < 3; ++i) {
        dot_product += h_result[i];
    }
    
    // Calculate magnitudes for normalization
    float mag_a = glm::length(a);
    float mag_b = glm::length(b);
    
    // Avoid division by zero
    if (mag_a < 1e-6f || mag_b < 1e-6f) {
        result = 0.0f;
    } else {
        result = glm::clamp(dot_product / (mag_a * mag_b), 0.0f, 1.0f);
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return result;
}

} // namespace sep::quantum