#include "hip/hip_runtime.h"
// Disable fpclassify functions that cause conflicts with CUDA internal headers
#define _DISABLE_FPCLASSIFY_FUNCTIONS 1
#define __CUDA_INCLUDE_COMPILER_INTERNAL_HEADERS 1

#include <hip/hip_runtime.h>

#include <cstdio>

#include "qbsa.cuh"

namespace sep::quantum {

__global__ void qbsa_kernel(sep::quantum::bitspace::QBSAParams params) {
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < params.num_probes) {
        if (params.probe_indices[tid] != params.expectations[tid]) {
            atomicAdd(&params.corrections[0], 1);
        }
    }
}

bool launch_qbsa_kernel(const sep::quantum::bitspace::QBSAParams& params) {
const dim3 block(256);
const dim3 grid((params.num_probes + block.x - 1) / block.x);

qbsa_kernel<<<grid, block>>>(params);

return hipGetLastError() == hipSuccess &&
hipDeviceSynchronize() == hipSuccess;
}

// Helper for CUDA error checking
inline void checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        // Simple error logging - could be enhanced later
        printf("CUDA error in QBSA kernel: %s\n", hipGetErrorString(result));
    }
}

} // namespace sep::quantum
