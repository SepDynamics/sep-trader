/*
 * Copyright (c) 2025 SEP Engine Contributors
 *
 * Implementation of CUDA functions in the sep::cuda namespace
 */

// Disable fpclassify functions that cause conflicts with CUDA internal headers
#define _DISABLE_FPCLASSIFY_FUNCTIONS 1
#define __CUDA_INCLUDE_COMPILER_INTERNAL_HEADERS 1

// CRITICAL: For CUDA compilation, apply comprehensive std::array protection
#include <hip/hip_runtime.h>

#include <array>
#include <string>
#include <vector>

#include "core/cuda_types.hpp"
#include "core/result_types.h"

namespace sep::cuda {

struct CudaCore::Impl {
    bool initialized_ = false;
    int current_device_ = -1;
    std::vector<hipDeviceProp_t> device_properties_;
    CudaMetrics current_metrics_;
};

CudaCore::CudaCore() : impl_(new Impl) {}
CudaCore::~CudaCore() = default;

CudaCore& CudaCore::instance() {
    static CudaCore inst;
    return inst;
}

Error CudaCore::initialize(int device_id) {
    hipError_t err = ::hipSetDevice(device_id);
    if (err != hipSuccess) {
        return Error(sep::SEPResult::CUDA_ERROR, hipGetErrorString(err));
    }

    int device_count = 0;
    err = ::hipGetDeviceCount(&device_count);
    if (err != hipSuccess) {
        return Error(sep::SEPResult::CUDA_ERROR, hipGetErrorString(err));
    }

    impl_->device_properties_.resize(device_count);
    for (int i = 0; i < device_count; ++i) {
        err = ::hipGetDeviceProperties(&impl_->device_properties_[i], i);
        if (err != hipSuccess) {
            return Error(sep::SEPResult::CUDA_ERROR, hipGetErrorString(err));
        }
    }

    impl_->initialized_ = true;
    impl_->current_device_ = device_id;
    return Error();
}

bool CudaCore::is_initialized() const {
    return impl_->initialized_;
}

Error CudaCore::setDevice(int device) {
    hipError_t err = ::hipSetDevice(device);
    if (err != hipSuccess) {
        return Error(sep::SEPResult::CUDA_ERROR, hipGetErrorString(err));
    }
    impl_->current_device_ = device;
    return Error();
}

int CudaCore::getDeviceCount() const {
    int count = 0;
    ::hipGetDeviceCount(&count);
    return count;
}

Error CudaCore::getDeviceProperties(hipDeviceProp_t& props, int device) const {
    hipError_t err = ::hipGetDeviceProperties(&props, device);
    if (err != hipSuccess) {
        return Error(sep::SEPResult::CUDA_ERROR, hipGetErrorString(err));
    }
    return Error();
}

Error CudaCore::getMemoryInfo(size_t& free, size_t& total) const {
    hipError_t err = ::hipMemGetInfo(&free, &total);
    if (err != hipSuccess) {
        return Error(sep::SEPResult::CUDA_ERROR, hipGetErrorString(err));
    }
    return Error();
}

Error CudaCore::getLastError() const {
    hipError_t err = ::hipGetLastError();
    if (err != hipSuccess) {
        return Error(sep::SEPResult::CUDA_ERROR, hipGetErrorString(err));
    }
    return Error();
}

std::string CudaCore::getErrorString(hipError_t error) const { return ::hipGetErrorString(error); }

CudaMetrics CudaCore::getMetrics() const {
    return impl_->current_metrics_;
}

Error CudaCore::updateMetrics() {
    size_t free_mem, total_mem;
    hipError_t err = ::hipMemGetInfo(&free_mem, &total_mem);
    if (err != hipSuccess) {
        return Error(sep::SEPResult::CUDA_ERROR, hipGetErrorString(err));
    }

    impl_->current_metrics_.total_memory = total_mem;
    impl_->current_metrics_.used_memory = total_mem - free_mem;
    impl_->current_metrics_.memory_utilization =
        (total_mem > 0) ? static_cast<float>(impl_->current_metrics_.used_memory) / total_mem : 0.0f;

    // GPU utilization would require NVML or similar API
    impl_->current_metrics_.gpu_utilization = 0.0f;

    return Error();
}

}  // namespace sep::cuda
