#include <hip/hip_runtime.h>

#include "event.h"
#include "stream.h"

namespace sep::cuda {

Event::Event(unsigned int flags) {
    hipEventCreateWithFlags(&event_, flags);
}

Event::~Event() {
    if (event_) {
        hipEventDestroy(event_);
    }
}

Event::Event(Event&& other) noexcept : event_(other.event_) {
    other.event_ = nullptr;
}

Event& Event::operator=(Event&& other) noexcept {
    if (this != &other) {
        if (event_) {
            hipEventDestroy(event_);
        }
        event_ = other.event_;
        other.event_ = nullptr;
    }
    return *this;
}

void Event::record(Stream& stream) {
    if (event_ && stream.isValid()) {
        hipEventRecord(event_, stream.handle());
    }
}

void Event::synchronize() {
    if (event_) {
        hipEventSynchronize(event_);
    }
}

float Event::elapsedTime(Event& start) {
    float time = 0.0f;
    if (event_ && start.valid()) {
        hipEventElapsedTime(&time, start.handle(), event_);
    }
    return time;
}

hipEvent_t Event::handle() const {
    return event_;
}

bool Event::valid() const {
    return event_ != nullptr;
}

}  // namespace sep::cuda
