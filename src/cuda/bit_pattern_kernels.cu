#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <algorithm> // For std::min, std::max
// Removed <cmath> include to fix fpclassify errors

#include "core/result_types.h"
#include "cuda/bit_pattern_types.cuh"  // Include the new device types

// Helper device functions (will be ported from forward_window_kernels.cpp)
__device__ bool detectTrendAcceleration(const uint8_t* window, size_t window_size);
__device__ bool detectMeanReversion(const uint8_t* window, size_t window_size);
__device__ bool detectVolatilityBreakout(const uint8_t* window, size_t window_size);

// CUDA kernel to analyze bit patterns
__global__ void analyzeBitPatternsKernel(const uint8_t* d_bits,
                                         size_t total_bits_size,
                                         size_t index_start,
                                         size_t window_size,
                                         sep::apps::cuda::ForwardWindowResultDevice* d_results) {
    // Each thread processes one window, but for simplicity, we'll assume one window for now
    // This kernel needs to be adapted for batch processing if multiple windows are to be processed in parallel.
    // For now, we'll assume a single window is passed and processed by thread 0.
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        sep::apps::cuda::ForwardWindowResultDevice result;
        result.flip_count = 0;
        result.rupture_count = 0;
        result.entropy = 0.0f;
        result.coherence = 0.0f;
        result.stability = 0.0f;
        result.confidence = 0.0f;

        if (total_bits_size <= index_start + 1) {
            d_results[0] = result;
            return;
        }

        // Create a local window for processing
        // This is a simplification; for larger windows, this would need to be optimized
        // (e.g., shared memory, or processing directly from global memory)
        uint8_t local_window[10]; // Assuming max window_size is 10 for now
        for (size_t i = 0; i < window_size; ++i) {
            local_window[i] = d_bits[index_start + i];
        }

        // Calculate flip and rupture counts
        for (size_t i = 1; i < window_size; ++i) {
            if (local_window[i-1] != local_window[i]) {
                result.flip_count++;
            } else if (local_window[i-1] == 1 && local_window[i] == 1) {
                result.rupture_count++;
            }
        }

        // Calculate entropy (Shannon entropy)
        size_t ones = 0;
        for (size_t i = 0; i < window_size; ++i) {
            if (local_window[i] == 1) ones++;
        }
        size_t zeros = window_size - ones;

        if (ones > 0 && zeros > 0) {
            double p1 = static_cast<double>(ones) / window_size;
            double p0 = static_cast<double>(zeros) / window_size;
            result.entropy = -(p1 * log2(p1) + p0 * log2(p0));
        } else {
            result.entropy = 0.0f;
        }

        // Basic coherence and stability estimates based on bit distribution
        float imbalance = fabsf(static_cast<float>(ones - zeros)) / window_size;
        result.coherence = 1.0f - imbalance;
        result.stability = 1.0f - result.entropy;

        // Set confidence based on window size and pattern consistency
        result.confidence = fminf(1.0f, static_cast<float>(window_size) / 10.0f) * result.coherence;

        d_results[0] = result;
    }
}

// Host-side launcher function
extern "C" sep::SEPResult launchAnalyzeBitPatternsKernel(const uint8_t* h_bits,
                                                      size_t total_bits_size,
                                                      size_t index_start,
                                                      size_t window_size,
                                                      sep::apps::cuda::ForwardWindowResultDevice* h_results,
                                                      hipStream_t stream) {
    uint8_t* d_bits = nullptr;
    sep::apps::cuda::ForwardWindowResultDevice* d_results = nullptr;
    sep::SEPResult result = sep::SEPResult::SUCCESS;

    hipError_t err = hipMallocAsync(&d_bits, total_bits_size * sizeof(uint8_t), stream);
    if (err != hipSuccess) {
        // Error handling now standardized with Result<T> pattern
        result = sep::SEPResult::CUDA_ERROR;
        goto cleanup;
    }

    err = hipMemcpyAsync(d_bits, h_bits, total_bits_size * sizeof(uint8_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) {
        // Error handling now standardized with Result<T> pattern
        result = sep::SEPResult::CUDA_ERROR;
        goto cleanup;
    }

    err = hipMallocAsync(&d_results, sizeof(sep::apps::cuda::ForwardWindowResultDevice), stream);
    if (err != hipSuccess) {
        // Error handling now standardized with Result<T> pattern
        result = sep::SEPResult::CUDA_ERROR;
        goto cleanup;
    }

    analyzeBitPatternsKernel<<<1, 1, 0, stream>>>(d_bits, total_bits_size, index_start, window_size, d_results);
    err = hipGetLastError();
    if (err != hipSuccess) {
        result = sep::SEPResult::CUDA_ERROR;
        goto cleanup;
    }

    err = hipMemcpyAsync(h_results, d_results, sizeof(sep::apps::cuda::ForwardWindowResultDevice), hipMemcpyDeviceToHost, stream);
    if (err != hipSuccess) {
        result = sep::SEPResult::CUDA_ERROR;
        goto cleanup;
    }

    err = hipStreamSynchronize(stream);
    if (err != hipSuccess) {
        result = sep::SEPResult::CUDA_ERROR;
        goto cleanup;
    }

cleanup:
    if (d_bits) hipFreeAsync(d_bits, stream);
    if (d_results) hipFreeAsync(d_results, stream);

    return result;
}
