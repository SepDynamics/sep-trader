#include <hip/hip_runtime.h>
#include <cstdint>

#include <iostream>
#include <stdexcept>
#include <cstdint>

#include "core/cuda_error.cuh"
#include "device_buffer.h"
#include "memory.h"

namespace sep {
namespace cuda {

// RAII wrapper implementation
template <typename T>
DeviceMemory<T>::DeviceMemory(size_t size) : size_(size) {
    if (size > 0) {
        hipError_t err = hipMalloc(&ptr_, size_ * sizeof(T));
        if (err != hipSuccess) {
            throw std::runtime_error(
                "Failed to allocate device memory: " +
                std::string(hipGetErrorString(err)));
        }
    }
}

template <typename T>
DeviceMemory<T>::~DeviceMemory() {
    if (ptr_) {
        hipFree(ptr_);
    }
}

template <typename T>
DeviceMemory<T>::DeviceMemory(DeviceMemory&& other) noexcept
    : ptr_(other.ptr_), size_(other.size_) {
    other.ptr_ = nullptr;
    other.size_ = 0;
}

template <typename T>
DeviceMemory<T>& DeviceMemory<T>::operator=(DeviceMemory&& other) noexcept {
    if (this != &other) {
        if (ptr_) {
            hipFree(ptr_);
        }
        ptr_ = other.ptr_;
        size_ = other.size_;
        other.ptr_ = nullptr;
        other.size_ = 0;
    }
    return *this;
}

template <typename T>
T* DeviceMemory<T>::get() const {
    return ptr_;
}

template <typename T>
size_t DeviceMemory<T>::size() const {
    return size_;
}

// Device memory utilities
void getDeviceMemoryInfo(size_t& free, size_t& total) {
    CUDA_CHECK(hipMemGetInfo(&free, &total));
}

void printDeviceMemoryStats() {
    size_t free_mem, total_mem;
    CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));

    std::cout << "CUDA Memory Usage:" << std::endl;
    std::cout << "  Total memory: " << (total_mem / (1024.0 * 1024.0)) << " MB" << std::endl;
    std::cout << "  Free memory:  " << (free_mem / (1024.0 * 1024.0)) << " MB" << std::endl;
    std::cout << "  Used memory:  " << ((total_mem - free_mem) / (1024.0 * 1024.0)) << " MB" << std::endl;
    std::cout << "  Usage:        " << ((total_mem - free_mem) * 100.0 / total_mem) << "%" << std::endl;
}

// Pinned memory
template <typename T>
T* allocatePinnedMemory(size_t count) {
    T* ptr = nullptr;
    CUDA_CHECK(hipHostMalloc(&ptr, count * sizeof(T)));
    return ptr;
}

template <typename T>
void freePinnedMemory(T* ptr) {
    if (ptr) {
        CUDA_CHECK(hipHostFree(ptr));
    }
}

// Managed memory
template <typename T>
T* allocateManagedMemory(size_t count) {
    T* ptr = nullptr;
    CUDA_CHECK(hipMallocManaged(&ptr, count * sizeof(T)));
    return ptr;
}

template <typename T>
void freeManagedMemory(T* ptr) {
    if (ptr) {
        CUDA_CHECK(hipFree(ptr));
    }
}

template <typename T>
void prefetchToDevice(T* ptr, size_t count, int device_id) {
    CUDA_CHECK(hipMemPrefetchAsync(ptr, count * sizeof(T), device_id));
}

template <typename T>
void prefetchToHost(T* ptr, size_t count) {
    CUDA_CHECK(hipMemPrefetchAsync(ptr, count * sizeof(T), hipCpuDeviceId));
}

template <typename T>
void zeroDeviceMemory(T* device_ptr, size_t count) {
    CUDA_CHECK(hipMemset(device_ptr, 0, count * sizeof(T)));
}

// Explicit template instantiations
template class DeviceMemory<float>;
template class DeviceMemory<double>;
template class DeviceMemory<int>;
template class DeviceMemory<char>;

template uint32_t* allocatePinnedMemory<uint32_t>(size_t);
template void freePinnedMemory<uint32_t>(uint32_t*);
template uint32_t* allocateManagedMemory<uint32_t>(size_t);
template void freeManagedMemory<uint32_t>(uint32_t*);
template void prefetchToDevice<uint32_t>(uint32_t*, size_t, int);
template void prefetchToHost<uint32_t>(uint32_t*, size_t);
template void zeroDeviceMemory<uint32_t>(uint32_t*, size_t);

template float* allocatePinnedMemory<float>(size_t);
template void freePinnedMemory<float>(float*);
template float* allocateManagedMemory<float>(size_t);
template void freeManagedMemory<float>(float*);
template void prefetchToDevice<float>(float*, size_t, int);
template void prefetchToHost<float>(float*, size_t);
template void zeroDeviceMemory<float>(float*, size_t);

template double* allocatePinnedMemory<double>(size_t);
template void freePinnedMemory<double>(double*);
template double* allocateManagedMemory<double>(size_t);
template void freeManagedMemory<double>(double*);
template void prefetchToDevice<double>(double*, size_t, int);
template void prefetchToHost<double>(double*, size_t);
template void zeroDeviceMemory<double>(double*, size_t);

} // namespace cuda
} // namespace sep

