#include "hip/hip_runtime.h"
#ifdef SEP_USE_CUDA
#include <hip/hip_runtime.h>
#endif

#include <cstdio>

#include "quantum/qbsa.cuh"

namespace sep::quantum {

__global__ void qbsa_kernel(QBSAParams params) {
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < params.num_probes) {
        if (params.probe_indices[tid] != params.expectations[tid]) {
            atomicAdd(&params.corrections[0], 1);
        }
    }
}

bool launch_qbsa_kernel(const QBSAParams& params) {
const dim3 block(256);
const dim3 grid((params.num_probes + block.x - 1) / block.x);

qbsa_kernel<<<grid, block>>>(params);

return hipGetLastError() == hipSuccess &&
hipDeviceSynchronize() == hipSuccess;
}

// Helper for CUDA error checking
inline void checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        // Simple error logging - could be enhanced later
        printf("CUDA error in QBSA kernel: %s\n", hipGetErrorString(result));
    }
}

} // namespace sep::quantum